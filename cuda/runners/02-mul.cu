#include "KernelMul.cuh"
#include <cstdlib>
#include <fstream>

void ExportData(const int p, const int block_size, const float time) {
    std::ofstream file;
    file.open("./02-mul.csv", std::ios::app);
    file << p << "," << block_size << "," << time << std::endl;
    file.close();
}

int main(int argc, char* argv[]) {

    int p = atoi(argv[1]);

    int N = 1 << p;
    int size = N * sizeof(float);
    float* h_x = (float*)malloc(size);
    float* h_y = (float*)malloc(size);
    float* h_result = (float*)malloc(size);

    float* d_x;
    float* d_y;
    float* d_result;

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_result, size);

    for (int i = 0; i < N; ++i) {
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    int blockSize = atoi(argv[2]);
    int numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    KernelMul<<<numBlocks, blockSize>>>(N, d_x, d_y, d_result);

    hipMemcpy(h_result, d_result, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    ExportData(p, blockSize, milliseconds);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
    free(h_x);
    free(h_y);
    free(h_result);
    return 0;
}
